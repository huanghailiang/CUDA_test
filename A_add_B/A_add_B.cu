#include "hip/hip_runtime.h"
#include "A_add_B.h"

int main(void)
{
	//���鸳��ֵ
	for(int i=0;i<N;i++)
	{
		A[i]=1;
		B[i]=1;
	}

	//��ʼCPU����
	start_CPU = clock();
	for(int i=0;i<N;i++)
	{
		C[i]=A[i]+B[i];
	}
	finish_CPU = clock();

	//��ʼ��GPU����Ĳ���
	init(N);
	
	//��ʼGPU����
	start_GPU = clock();
	//*vector_add(A,B,N);
	//ִ�к˺���
	AaddB<<<num_blocks, threadsPerBlock>>>(dev_A, dev_B, dev_C, N);
	finish_GPU = clock();
printf("C[10]=%10f      \n",C[0]);

	// �����������
	hipMemcpy(C , dev_C, size, hipMemcpyDeviceToHost);

	//�ͷ��ڴ�
	hipFree(dev_A);
	hipFree(dev_B);
	hipFree(dev_C);

	//�������鿴
	for(int i=0;i<N;i++)
	{
		//printf("A=%f   %d\n",A[i],i );
		//printf("B=%f   %d\n",B[i],i );
		//printf("%f+%f=%f   %d\n",A[i],B[i],C[i],i );
	}
	printf("\n");
	//��ʾ�̺߳��߳̿������
	printf("threadsPerBlock =%d\n",threadsPerBlock  );
	printf("num_blocks=%d\n",num_blocks);
	printf("\n");
	//��ʾGPU����ʱ
	time_GPU = (double)(finish_GPU - start_GPU)*1000 ;
	printf( "time_GPU=%f us\n", time_GPU);
	printf("\n");
	//��ʾCPU����ʱ
	time_CPU = (double)(finish_CPU - start_CPU)*1000 ;
	printf( "time_CPU=%f us\n", time_CPU);

	return 0;
}