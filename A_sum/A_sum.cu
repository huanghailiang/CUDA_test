#include "hip/hip_runtime.h"
#include "A_sum.h"

int main()
{
	//���鸳ֵ
	for(int i=0;i<N;i++)
	{
		A[i]=i+1;
	}

	//��ʼCPU����
	start_CPU = clock();
	for(int i=0;i<N;i++)
	{
		sum+=i+1;
	}
	finish_CPU = clock();

	//�ڴ�ȳ�ʼ��
	init (N);

	//��ʼGPU����
	start_GPU = clock();

	//square_sum_array(A,N);
	A_sum<<<num_blocks, threads_per_block, threads_per_block*sizeof(float)>>>(sum_a, dsum_per_block, N);
	//����GPU�ڴ浽CPU�ڴ�
	hipMemcpy(sum_per_block, dsum_per_block, (num_blocks+1)*sizeof(float), hipMemcpyDeviceToHost);
	  for( int i=0; i<num_blocks; i++ )
	{
		//printf("sum_per_block=%f  %d\n",sum_per_block[i],i);
		sum_all+=sum_per_block[i];
	}

	finish_GPU = clock();
	//�ͷ��ڴ�
	hipFree(dsum_per_block);
	hipFree(sum_a);
	free(sum_per_block);

	printf("\n");
	
	//�������鿴
	printf ("GPU:sum_all=%f\n",sum_all);
	printf ("CPU:sum=%f\n",(float)sum);	

	//��ʾ�̺߳��߳̿������
	//printf("threadsPerBlock =%d\n",threadsPerBlock  );
	//printf("num_blocks=%d\n",num_blocks);
	printf("\n");
	//��ʾGPU����ʱ
	time_GPU = (double)(finish_GPU - start_GPU)*1000 ;
	printf( "time_GPU=%f us\n", time_GPU);	
	//��ʾCPU����ʱ
	time_CPU = (double)(finish_CPU - start_CPU)*1000 ;
	printf( "time_CPU=%f us\n", time_CPU);
	printf("\n");

	return 0;
}