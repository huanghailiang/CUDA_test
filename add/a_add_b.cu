#include "a_add_b.h"

int main (void)
{
	hipMalloc((void **)&dev_c,sizeof(int));
	add<<<1,1>>>(2,7,dev_c);
	hipMemcpy(&c,dev_c,sizeof(int),hipMemcpyDeviceToHost);
	printf("2+7=%d\n",c);
	hipFree(dev_c);
	return 0;
}